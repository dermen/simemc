#include "hip/hip_runtime.h"

#include <mpi.h>
#include "emc_ext.h"
#include "general.cuh"

__global__
void orientMultiply(VEC3*  qVecs, MAT3* rotMats, int numQ,
                    int numRot, bool* out, CUDAREAL hcut,
                    int minPred, MAT3 Bmat);

void set_blocks_threads(gpuOrient& gpu, int numRot);

void free_orientMatch(gpuOrient& gpu){
    if (gpu.rotMats != NULL && gpu.free_rotMats)
        gpuErr(hipFree(gpu.rotMats));
    if(gpu.qVecs != NULL)
        gpuErr(hipFree(gpu.qVecs));
    if (gpu.out != NULL)
        gpuErr(hipFree(gpu.out));
    if (gpu.close_rotMats_handle)
        gpuErr(hipIpcCloseMemHandle(gpu.rotMats));
}


void set_blocks_threads(gpuOrient& gpu, int numRot){
    // optional size of each device block, else default to 128
    char* diffBragg_threads = getenv("ORIENT_THREADS_PER_BLOCK");
    if (diffBragg_threads==NULL)
        gpu.blockSize=128;
    else
        gpu.blockSize=atoi(diffBragg_threads);
    gpu.numBlocks = (numRot+gpu.blockSize-1)/gpu.blockSize;
}

void setup_orientMatch_IPC(int dev_id, int maxNumQ, gpuOrient& gpu,
                       np::ndarray& Umats, int numRot, MPI_Comm COMM){

    int rank, size;
    MPI_Comm_size(COMM, &size);
    MPI_Comm_rank(COMM, &rank);

    gpuErr(hipSetDevice(dev_id));
    gpu.numRot = numRot;
    gpu.max_numQ = maxNumQ;
    gpu.device = dev_id;
    gpuErr(hipMallocManaged((void **)&gpu.out, numRot*sizeof(bool)));
    gpuErr(hipMallocManaged((void **)&gpu.qVecs, maxNumQ*sizeof(VEC3)));

    hipIpcMemHandle_t rotMats_memHand;//[1];
    if (rank==0){
        get_mem_handle(rotMats_memHand, gpu.rotMats, Umats, numRot);
    }
    // broadcast and copy the memoryhandle to gpu.rotMats on other processes
    broadcast_ipc_handle(rotMats_memHand, gpu.rotMats, COMM);

    gpu.free_rotMats = (rank==0) ;
    gpu.close_rotMats_handle = (rank >0) ;

    set_blocks_threads(gpu, numRot);
}


void setup_orientMatch(int dev_id, int maxNumQ, gpuOrient& gpu,
                       np::ndarray& Umats, bool alloc ){
    int numRot = Umats.shape(0)/9;
    if (alloc){
        gpu.numRot = numRot;
        gpu.max_numQ = maxNumQ;
        gpuErr(hipSetDevice(dev_id));
        gpu.device = dev_id;
        gpuErr(hipMallocManaged((void **)&gpu.rotMats, numRot*sizeof(MAT3)));
        gpuErr(hipMallocManaged((void **)&gpu.out, numRot*sizeof(bool)));
        gpuErr(hipMallocManaged((void **)&gpu.qVecs, maxNumQ*sizeof(VEC3)));
    }
    copy_umats(gpu.rotMats, Umats, numRot);
    set_blocks_threads(gpu, numRot);
}


void orientPeaks(gpuOrient& gpu, np::ndarray& qvecs, CUDAREAL hcut,
                 int minPred, bool verbose){

    double time;
    struct timeval t1, t2;//, t3 ,t4;

    gettimeofday(&t1, 0);
    int numQ = qvecs.shape(0)/3;

    if (verbose)
        printf("Setting cuda device %d\n", gpu.device);
    gpuErr(hipSetDevice(gpu.device));
    if (numQ > gpu.max_numQ){
        printf("WARNING: re-allocating because maximum num Q vecs was exceeded!! Now maxNumQ =%d (was %d)\n",
               numQ, gpu.max_numQ);
        gpu.max_numQ = numQ;
        if (gpu.qVecs != NULL)
        gpuErr(hipFree(gpu.qVecs));
        gpuErr(hipMallocManaged((void **)&gpu.qVecs, gpu.max_numQ*sizeof(VEC3)));
    }

    // copy the Qvectors to the device
    if (verbose)
        printf("Copying over %d qvectors to the GPU\n", numQ);
    CUDAREAL* qptr = reinterpret_cast<CUDAREAL*>(qvecs.get_data());
    for (int i_q=0; i_q < numQ; i_q++){
        int i = i_q*3;
        CUDAREAL qx = *(qptr+i);
        CUDAREAL qy = *(qptr+i+1);
        CUDAREAL qz = *(qptr+i+2);
        VEC3 Q(qx,qy,qz);
        gpu.qVecs[i_q] = Q;
    }
    gettimeofday(&t2, 0);
    time = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000.0;
    if(verbose)
        printf("Pre-kernel time=%f msec\n", time);

    gettimeofday(&t1, 0);
    // run the kernel
    MAT3 Binv = gpu.Bmat.inverse();
    orientMultiply<<<gpu.numBlocks, gpu.blockSize>>>
        (gpu.qVecs, gpu.rotMats, numQ, gpu.numRot,
         gpu.out, hcut, minPred, Binv);

    error_msg(hipGetLastError(), gpu.mpi_rank);
    hipDeviceSynchronize();
    gettimeofday(&t2, 0);
    time = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000.0;
    if(verbose)
        printf("kernel time=%f msec\n", time);

}


__global__
void orientMultiply(VEC3* qVecs, MAT3* rotMats, int numQ,
                    int numRot, bool* out, CUDAREAL hcut,
                    int minPred, MAT3 Bmat){

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int thread_stride = blockDim.x * gridDim.x;

    for (int i_rot=tid; i_rot < numRot; i_rot += thread_stride){
        int count=0;
        for (int i_q=0; i_q < numQ; i_q ++ ){
            VEC3 Hkl = Bmat*(rotMats[i_rot]*qVecs[i_q]);

            CUDAREAL h = ceil(Hkl[0]-0.5);
            CUDAREAL k = ceil(Hkl[1]-0.5);
            CUDAREAL l = ceil(Hkl[2]-0.5);
            VEC3 Hi(h,k,l);
            VEC3 deltaH = Hkl-Hi;
            CUDAREAL hnorm = deltaH.norm();
            if (hnorm < hcut)
                count += 1;
        }
        if (count >= minPred)
            out[i_rot] = true;
        else
            out[i_rot] = false;
    }
}

