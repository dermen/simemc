#include "hip/hip_runtime.h"

#include <hipcub/hipcub.hpp>
#include "cuda_trilerp.h"
#include "emc.cuh"

__device__ __inline__ unsigned int get_densities_index(int i,int j,int k, int nx, int ny, int nz);

__global__ void trilinear_interpolation_rotate_on_GPU(const CUDAREAL* __restrict__ densities,
                                        VEC3*vectors, CUDAREAL* out, MAT3 rotMat,
                                        int num_qvec,
                                        int nx, int ny, int nz,
                                        CUDAREAL cx, CUDAREAL cy, CUDAREAL cz,
                                        CUDAREAL dx, CUDAREAL dy, CUDAREAL dz);

__global__ void trilinear_insertion_rotate_on_GPU(
        CUDAREAL * densities,
        CUDAREAL * wts,
        CUDAREAL* insertion_values,
        CUDAREAL tomo_wt,
        VEC3 *vectors,
        MAT3 rotMat, int num_qvec,
        int nx, int ny, int nz,
        CUDAREAL cx, CUDAREAL cy, CUDAREAL cz,
        CUDAREAL dx, CUDAREAL dy, CUDAREAL dz);


__global__ void EMC_equation_two(const CUDAREAL* __restrict__ densities,
                                 const CUDAREAL* __restrict__ data,
                                 CUDAREAL scale_factor,
                                 VEC3*vectors, CUDAREAL* out_rot,
                                 MAT3* rotMats, int * rot_inds,
                                 int numRot, int num_qvec,
                                 int nx, int ny, int nz,
                                 CUDAREAL cx, CUDAREAL cy, CUDAREAL cz,
                                 CUDAREAL dx, CUDAREAL dy, CUDAREAL dz,
                                 const bool compute_derivative);


void prepare_for_lerping(lerpy& gpu, np::ndarray Umats, np::ndarray densities, 
                        np::ndarray qvectors){
    gpu.numRot = Umats.shape(0)/9;
    gpu.numQ = qvectors.shape(0)/3;
    // TODO global verbose flag
    //printf("Number of Qvectors=%d\n", gpu.numQ);
    gpu.numDens = densities.shape(0);
   // TODO asserts on len of corner and delta (must be 3)

    gpuErr(hipSetDevice(gpu.device));
    gpuErr(hipMallocManaged((void **)&gpu.rotMats, gpu.numRot*sizeof(MAT3)));
    gpuErr(hipMallocManaged((void **)&gpu.densities, gpu.numDens*sizeof(CUDAREAL)));
    gpuErr(hipMallocManaged((void **)&gpu.out, gpu.maxNumQ*sizeof(CUDAREAL)));
    gpuErr(hipMallocManaged((void **)&gpu.out_equation_two, gpu.maxNumRotInds*sizeof(CUDAREAL)));
    gpuErr(hipMallocManaged((void **)&gpu.qVecs, gpu.maxNumQ*sizeof(VEC3)));
    gpuErr(hipMallocManaged((void **)&gpu.rotInds, gpu.maxNumRotInds*sizeof(int)));
    gpuErr(hipMallocManaged((void **)&gpu.data, gpu.numDataPixels*sizeof(CUDAREAL)));

    MAT3 Umat; // orientation matrix
    CUDAREAL* Umats_ptr = reinterpret_cast<CUDAREAL*>(Umats.get_data());
    for (int i_rot=0; i_rot < gpu.numRot; i_rot ++){
        int i= i_rot*9;
        CUDAREAL uxx = *(Umats_ptr+i);
        CUDAREAL uxy = *(Umats_ptr+i+1);
        CUDAREAL uxz = *(Umats_ptr+i+2);
        CUDAREAL uyx = *(Umats_ptr+i+3);
        CUDAREAL uyy = *(Umats_ptr+i+4);
        CUDAREAL uyz = *(Umats_ptr+i+5);
        CUDAREAL uzx = *(Umats_ptr+i+6);
        CUDAREAL uzy = *(Umats_ptr+i+7);
        CUDAREAL uzz = *(Umats_ptr+i+8);
        Umat << uxx, uxy, uxz,
                uyx, uyy, uyz,
                uzx, uzy, uzz;
        gpu.rotMats[i_rot] = Umat.transpose();
    }

    CUDAREAL* qvec_ptr = reinterpret_cast<CUDAREAL*>(qvectors.get_data());
    for (int i_q = 0; i_q < gpu.numQ; i_q++) {
        int i = i_q * 3;
        CUDAREAL qx = *(qvec_ptr +i);
        CUDAREAL qy = *(qvec_ptr +i+1);
        CUDAREAL qz = *(qvec_ptr +i+2);
        VEC3 Q(qx, qy, qz);
        gpu.qVecs[i_q] = Q;
    }

    CUDAREAL* dens_ptr = reinterpret_cast<CUDAREAL*>(densities.get_data());
    for (int i=0; i < gpu.numDens; i++){
        gpu.densities[i] = *(dens_ptr+i);
    }
}

void shot_data_to_device(lerpy& gpu, np::ndarray& shot_data){
    unsigned int num_pix = shot_data.shape(0);
    CUDAREAL* data_ptr = reinterpret_cast<CUDAREAL*>(shot_data.get_data());
    for (int i=0; i < num_pix; i++) {
        gpu.data[i] = *(data_ptr + i);
    }
}

void densities_to_device(lerpy& gpu, np::ndarray& new_densities){
    unsigned int numDens = new_densities.shape(0);
    CUDAREAL* dens_ptr = reinterpret_cast<CUDAREAL*>(new_densities.get_data());
    for (int i=0; i < gpu.numDens; i++){
        gpu.densities[i] = *(dens_ptr+i);
    }
}

void toggle_insert_mode(lerpy& gpu){
    if (gpu.wts==NULL){
        gpuErr(hipMallocManaged((void **)&gpu.wts, gpu.numDens*sizeof(CUDAREAL)));
    }

    for (int i=0; i < gpu.numDens; i++){
        gpu.wts[i]=0;
        gpu.densities[i]=0;
    }
}


void do_a_lerp(lerpy& gpu, std::vector<int>& rot_inds, bool verbose, int task) {
    double time;
    struct timeval t1, t2;//, t3 ,t4;

    gettimeofday(&t1, 0);

    // optional size of each device block, else default to 128
    char *threads = getenv("ORIENT_THREADS_PER_BLOCK");
    if (threads == NULL)
        gpu.blockSize = 128;
    else
        gpu.blockSize = atoi(threads);
    //gpu.blockSize=blockSize;
    gpu.numBlocks = (gpu.numQ + gpu.blockSize - 1) / gpu.blockSize;

    int numRotInds = rot_inds.size();
    for (int i=0; i< numRotInds; i++){
        gpu.rotInds[i] = rot_inds[i];
        if(task==1 || task==3){
            gpu.out_equation_two[i] = 0;
        }
    }
    if (verbose) {
        gettimeofday(&t2, 0);
        time = (1000000.0 * (t2.tv_sec - t1.tv_sec) + t2.tv_usec - t1.tv_usec) / 1000.0;
        printf("Pre-kernel time=%f msec\n", time);
    }

    gettimeofday(&t1, 0);

    /*
     * KERNELS
     */
    if (task==0){
        MAT3 rotMat = gpu.rotMats[gpu.rotInds[0]];
        trilinear_interpolation_rotate_on_GPU<<<gpu.numBlocks, gpu.blockSize>>>
                (gpu.densities, gpu.qVecs, gpu.out,
                 rotMat, gpu.numQ,
                 256, 256, 256,
                 gpu.corner[0], gpu.corner[1], gpu.corner[2],
                 gpu.delta[0], gpu.delta[1], gpu.delta[2]
                );
    }
    else if(task==1 || task==3) {
        if (verbose)printf("Running equation 2!\n");
        EMC_equation_two<<<gpu.numBlocks, gpu.blockSize>>>
                (gpu.densities,  gpu.data, gpu.shot_scale,
                 gpu.qVecs, gpu.out_equation_two,
                 gpu.rotMats, gpu.rotInds, numRotInds, gpu.numQ,
                 256, 256, 256,
                 gpu.corner[0], gpu.corner[1], gpu.corner[2],
                 gpu.delta[0], gpu.delta[1], gpu.delta[2], task==3
                );

    }
    else if (task==2)  {
        if (verbose)printf("Trilinear insertion!\n");
        MAT3 rotMat = gpu.rotMats[gpu.rotInds[0]];
//      NOTE: here gpu.data are the insert values
        trilinear_insertion_rotate_on_GPU<<<gpu.numBlocks, gpu.blockSize>>>
                (gpu.densities, gpu.wts, gpu.data, gpu.tomogram_wt, gpu.qVecs,
                 rotMat, gpu.numQ,
                 256, 256, 256,
                 gpu.corner[0], gpu.corner[1], gpu.corner[2],
                 gpu.delta[0], gpu.delta[1], gpu.delta[2]
                );
   
    }
    //else {
    //    // TODO remove this block of code, dont think its used ...
    //    printf("Symmetrize density!\n");
    //    // zero out the densities
    //    toggle_insert_mode(gpu);

    //    MAT3 rotMat = gpu.rotMats[gpu.rotInds[0]];
//  //    NOTE: here gpu.data are the insert values
    //    trilinear_insertion_rotate_on_GPU<<<gpu.numBlocks, gpu.blockSize>>>
    //            (gpu.densities, gpu.wts, gpu.data, gpu.qVecs,
    //             rotMat, gpu.numQ,
    //             256, 256, 256,
    //             gpu.corner[0], gpu.corner[1], gpu.corner[2],
    //             gpu.delta[0], gpu.delta[1], gpu.delta[2]
    //            );
    //}
    error_msg(hipGetLastError(), "after kernel call");
    hipDeviceSynchronize();
    if (verbose) {
        gettimeofday(&t2, 0);
        time = (1000000.0 * (t2.tv_sec - t1.tv_sec) + t2.tv_usec - t1.tv_usec) / 1000.0;
        printf("kernel time=%f msec\n", time);
    }

    gettimeofday(&t1, 0);
    if (task==1 || task==3){
        bp::list outList;
        for (int i = 0; i < numRotInds; i++)
            outList.append(gpu.out_equation_two[i]);
        gpu.outList = outList;
    }
        
    if (verbose){
        gettimeofday(&t2, 0);
        time = (1000000.0 * (t2.tv_sec - t1.tv_sec) + t2.tv_usec - t1.tv_usec) / 1000.0;
        printf("Post-kernel time=%f msec\n", time);
    }
}

void free_lerpy(lerpy& gpu){
    if (gpu.qVecs != NULL)
        gpuErr(hipFree(gpu.qVecs));
    if (gpu.rotInds!= NULL)
        gpuErr(hipFree(gpu.rotInds));
    if (gpu.rotMats!= NULL)
        gpuErr(hipFree(gpu.rotMats));
    if (gpu.data!= NULL)
        gpuErr(hipFree(gpu.data));
    if (gpu.out!= NULL)
        gpuErr(hipFree(gpu.out));
    if (gpu.out_equation_two!= NULL)
        gpuErr(hipFree(gpu.out_equation_two));
    if (gpu.densities!= NULL)
        gpuErr(hipFree(gpu.densities));
    if (gpu.wts!= NULL)
        gpuErr(hipFree(gpu.wts));
}

__device__ __inline__ unsigned int get_densities_index(int i,int j,int k, int nx, int ny, int nz)
{
    //int idx = i + j*nx + k*nx*ny;
    unsigned int idx = fma(nx, fma(k,ny,j), i);
    return idx;
}

/**
 * this is mostly a CUDA port of the reborn trilinear interpolator written in Fortran:
 *     https://gitlab.com/kirianlab/reborn/-/blob/master/reborn/fortran/density.f90#L16
 */
__global__ void trilinear_interpolation_rotate_on_GPU(
                                        const CUDAREAL * __restrict__ densities, 
                                        VEC3 *vectors, CUDAREAL * out,
                                        MAT3 rotMat, int num_qvec,
                                        int nx, int ny, int nz,
                                        CUDAREAL cx, CUDAREAL cy, CUDAREAL cz,
                                        CUDAREAL dx, CUDAREAL dy, CUDAREAL dz){

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int thread_stride = blockDim.x * gridDim.x;
    CUDAREAL i_f, j_f, k_f;
    CUDAREAL x0,x1,y0,y1,z0,z1;
    CUDAREAL qx,qy,qz;
    int i0, i1, j0, j1, k0, k1;
    CUDAREAL I0,I1,I2,I3,I4,I5,I6,I7;
    CUDAREAL a0,a1,a2,a3,a4,a5,a6,a7;
    CUDAREAL x0y0, x1y1, x0y1, x1y0;
    int i;
    VEC3 Q;
    
    for (i=tid; i < num_qvec; i += thread_stride){
        Q = rotMat*vectors[i];
        qx = Q[0];
        qy = Q[1];
        qz = Q[2];

        k_f = (qx - cx) / dx;
        j_f = (qy - cy) / dy;
        i_f = (qz - cz) / dz;
        i0 = int(floor(i_f));
        j0 = int(floor(j_f));
        k0 = int(floor(k_f));
        if (i0 > nz-2 || j0 > ny-2 || k0 > nx-2 )
            continue;
        if(i0 < 0 || j0  < 0 || k0 < 0)
            continue;
        i1 = i0 + 1;
        j1 = j0 + 1;
        k1 = k0 + 1;

        x0 = i_f - i0;
        y0 = j_f - j0;
        z0 = k_f - k0;
        x1 = 1.0 - x0;
        y1 = 1.0 - y0;
        z1 = 1.0 - z0;

        I0 = __ldg(&densities[get_densities_index(i0, j0, k0, nx, ny, nz)]);
        I1 = __ldg(&densities[get_densities_index(i1, j0, k0, nx, ny, nz)]);
        I2 = __ldg(&densities[get_densities_index(i0, j1, k0, nx, ny, nz)]);
        I3 = __ldg(&densities[get_densities_index(i0, j0, k1, nx, ny, nz)]);
        I4 = __ldg(&densities[get_densities_index(i1, j0, k1, nx, ny, nz)]);
        I5 = __ldg(&densities[get_densities_index(i0, j1, k1, nx, ny, nz)]);
        I6 = __ldg(&densities[get_densities_index(i1, j1, k0, nx, ny, nz)]);
        I7 = __ldg(&densities[get_densities_index(i1, j1, k1, nx, ny, nz)]);

        x0y0 = x0*y0;
        x1y1 = x1*y1;
        x1y0 = x1*y0;
        x0y1 = x0*y1;

        a0 = x1y1 * z1;
        a1 = x0y1 * z1;
        a2 = x1y0 * z1;
        a3 = x1y1 * z0;
        a4 = x0y1 * z0;
        a5 = x1y0 * z0;
        a6 = x0y0 * z1;
        a7 = x0y0 * z0;

        out[i] = I0 * a0 +
                 I1 * a1 +
                 I2 * a2 +
                 I3 * a3 +
                 I4 * a4 +
                 I5 * a5 +
                 I6 * a6 +
                 I7 * a7;
    }
}

/**
 * Insert a tomogram into the density
 *
 * this is mostly a CUDA port of the reborn trilinear insertion written in Fortran:
 *     https://gitlab.com/kirianlab/reborn/-/blob/master/reborn/fortran/density.f90#L16
 */
__global__ void trilinear_insertion_rotate_on_GPU(
        CUDAREAL*  densities,
        CUDAREAL*  wts,
        CUDAREAL* insertion_values,
        CUDAREAL tomo_wt,
        VEC3 *vectors,
        MAT3 rotMat, int num_qvec,
        int nx, int ny, int nz,
        CUDAREAL cx, CUDAREAL cy, CUDAREAL cz,
        CUDAREAL dx, CUDAREAL dy, CUDAREAL dz){

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int thread_stride = blockDim.x * gridDim.x;
    CUDAREAL i_f, j_f, k_f;
    CUDAREAL x0,x1,y0,y1,z0,z1;
    CUDAREAL qx,qy,qz;
    int i0, i1, j0, j1, k0, k1;
    CUDAREAL a0,a1,a2,a3,a4,a5,a6,a7;
    CUDAREAL x0y0, x1y1, x0y1, x1y0;
    int i;

    VEC3 Q;
    int idx0,idx1,idx2,idx3,idx4,idx5,idx6,idx7;
    CUDAREAL val;
    for (i=tid; i < num_qvec; i += thread_stride){
        val = insertion_values[i];
        Q = rotMat*vectors[i];
        qx = Q[0];
        qy = Q[1];
        qz = Q[2];

        k_f = (qx - cx) / dx;
        j_f = (qy - cy) / dy;
        i_f = (qz - cz) / dz;
        i0 = int(floor(i_f));
        j0 = int(floor(j_f));
        k0 = int(floor(k_f));
        if (i0 > nz-2 || j0 > ny-2 || k0 > nx-2 )
            continue;
        if(i0 < 0 || j0  < 0 || k0 < 0)
            continue;
        i1 = i0 + 1;
        j1 = j0 + 1;
        k1 = k0 + 1;

        x0 = i_f - i0;
        y0 = j_f - j0;
        z0 = k_f - k0;
        x1 = 1.0 - x0;
        y1 = 1.0 - y0;
        z1 = 1.0 - z0;

        x0y0 = x0*y0;
        x1y1 = x1*y1;
        x1y0 = x1*y0;
        x0y1 = x0*y1;

        z1 *= tomo_wt;
        z0 *= tomo_wt;

        a0 = x1y1 * z1;
        a1 = x0y1 * z1;
        a2 = x1y0 * z1;
        a3 = x1y1 * z0;
        a4 = x0y1 * z0;
        a5 = x1y0 * z0;
        a6 = x0y0 * z1;
        a7 = x0y0 * z0;
        idx0 = get_densities_index(i0, j0, k0, nx, ny, nz);
        idx1 = get_densities_index(i1, j0, k0, nx, ny, nz);
        idx2 = get_densities_index(i0, j1, k0, nx, ny, nz);
        idx3 = get_densities_index(i0, j0, k1, nx, ny, nz);
        idx4 = get_densities_index(i1, j0, k1, nx, ny, nz);
        idx5 = get_densities_index(i0, j1, k1, nx, ny, nz);
        idx6 = get_densities_index(i1, j1, k0, nx, ny, nz);
        idx7 = get_densities_index(i1, j1, k1, nx, ny, nz);

        atomicAdd(&densities[idx0], val*a0);
        atomicAdd(&densities[idx1], val*a1);
        atomicAdd(&densities[idx2], val*a2);
        atomicAdd(&densities[idx3], val*a3);
        atomicAdd(&densities[idx4], val*a4);
        atomicAdd(&densities[idx5], val*a5);
        atomicAdd(&densities[idx6], val*a6);
        atomicAdd(&densities[idx7], val*a7);

        atomicAdd(&wts[idx0], a0);
        atomicAdd(&wts[idx1], a1);
        atomicAdd(&wts[idx2], a2);
        atomicAdd(&wts[idx3], a3);
        atomicAdd(&wts[idx4], a4);
        atomicAdd(&wts[idx5], a5);
        atomicAdd(&wts[idx6], a6);
        atomicAdd(&wts[idx7], a7);
    }
}

/*
 * Computes equation (2) in http://dx.doi.org/10.1107/S1600576716008165
 *
 */
__global__ void EMC_equation_two(const CUDAREAL * __restrict__ densities,
                                 const CUDAREAL * __restrict__ data,
                                 CUDAREAL scale_factor,
                                 VEC3 *vectors,
                                 CUDAREAL * out_rot,
                                 MAT3* rotMats, int* rot_inds, int numRot, int num_qvec,
                                 int nx, int ny, int nz,
                                 CUDAREAL cx, CUDAREAL cy, CUDAREAL cz,
                                 CUDAREAL dx, CUDAREAL dy, CUDAREAL dz,
                                 const bool compute_derivative){

    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int thread_stride = blockDim.x * gridDim.x;
    CUDAREAL i_f, j_f, k_f;
    CUDAREAL x0,x1,y0,y1,z0,z1;
    CUDAREAL qx,qy,qz;
    int i0, i1, j0, j1, k0, k1;
    CUDAREAL I0,I1,I2,I3,I4,I5,I6,I7;
    CUDAREAL a0,a1,a2,a3,a4,a5,a6,a7;
    CUDAREAL x0y0, x1y1, x0y1, x1y0;
    int i_rot;

    VEC3 Q;
    CUDAREAL K_t;
    CUDAREAL W_rt;
    int t,r;

    CUDAREAL R_dr_thread, R_dr_block;
    typedef hipcub::BlockReduce<CUDAREAL, 128> BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage;
    MAT3 R;

    for (i_rot =0; i_rot < numRot; i_rot++){
        R_dr_thread = 0;
        r = rot_inds[i_rot];
        R = rotMats[r];
        for (t=tid; t < num_qvec; t += thread_stride){
            K_t = __ldg(&data[t]);
            Q = R*vectors[t];
            qx = Q[0];
            qy = Q[1];
            qz = Q[2];

            k_f = (qx - cx) / dx;
            j_f = (qy - cy) / dy;
            i_f = (qz - cz) / dz;
            i0 = int(floor(i_f));
            j0 = int(floor(j_f));
            k0 = int(floor(k_f));
            if (i0 > nz-2 || j0 > ny-2 || k0 > nx-2 )
                continue;
            if(i0 < 0 || j0  < 0 || k0 < 0)
                continue;
            i1 = i0 + 1;
            j1 = j0 + 1;
            k1 = k0 + 1;

            x0 = i_f - i0;
            y0 = j_f - j0;
            z0 = k_f - k0;
            x1 = 1.0 - x0;
            y1 = 1.0 - y0;
            z1 = 1.0 - z0;

            I0 = __ldg(&densities[get_densities_index(i0, j0, k0, nx, ny, nz)]);
            I1 = __ldg(&densities[get_densities_index(i1, j0, k0, nx, ny, nz)]); 
            I2 = __ldg(&densities[get_densities_index(i0, j1, k0, nx, ny, nz)]); 
            I3 = __ldg(&densities[get_densities_index(i0, j0, k1, nx, ny, nz)]); 
            I4 = __ldg(&densities[get_densities_index(i1, j0, k1, nx, ny, nz)]); 
            I5 = __ldg(&densities[get_densities_index(i0, j1, k1, nx, ny, nz)]); 
            I6 = __ldg(&densities[get_densities_index(i1, j1, k0, nx, ny, nz)]); 
            I7 = __ldg(&densities[get_densities_index(i1, j1, k1, nx, ny, nz)]); 

            x0y0 = x0*y0;
            x1y1 = x1*y1;
            x1y0 = x1*y0;
            x0y1 = x0*y1;
           
            a0 = x1y1 * z1;
            a1 = x0y1 * z1;
            a2 = x1y0 * z1;
            a3 = x1y1 * z0;
            a4 = x0y1 * z0;
            a5 = x1y0 * z0;
            a6 = x0y0 * z1;
            a7 = x0y0 * z0;

            W_rt = fma(I0,a0,
                   fma(I1,a1,
                   fma(I2,a2,
                   fma(I3,a3,
                   fma(I4,a4,
                   fma(I5,a5,
                   fma(I6,a6,
                   fma(I7,a7,0))))))));
            if (compute_derivative) {
                if (W_rt > 0)
                    R_dr_thread += K_t/scale_factor - W_rt;
            }
            else{
                //W_rt *= scale_factor;
                if (W_rt > 0)
                    R_dr_thread += K_t * log(W_rt*scale_factor) - W_rt*scale_factor;
            }

        }
        __syncthreads();
        // reduce R_dr across blocks, store result on thread 0
        R_dr_block = BlockReduce(temp_storage).Sum(R_dr_thread);

        // accumulate across all thread 0 using atomics
        if (threadIdx.x==0)
            atomicAdd(&out_rot[i_rot], R_dr_block);
    }
}
